#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <gmp.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#include <chrono>

#include "../include/cgbn/cgbn.h"
#include "support.h"

#define SM2     // enable optimization for SM2 (a=-3)
// #define BIT256  // use 256-bit integer instead of 512-bit

// The CGBN context uses the following three parameters:
//   TBP             - threads per block (zero means to use the blockDim.x)
//   MAX_ROTATION    - must be small power of 2, imperically, 4 works well
//   SHM_LIMIT       - number of bytes of dynamic shared memory available to the kernel
//   CONSTANT_TIME   - require constant time algorithms (currently, constant time algorithms are not available)

// Locally it will also be helpful to have several parameters:
//   TPI             - threads per instance
//   BITS            - number of bits per instance

template <uint32_t tpi, uint32_t bits>
class gsv_params_t {
   public:
    // parameters used by the CGBN context
    static const uint32_t TPB = 0;            // get TPB from blockDim.x
    static const uint32_t MAX_ROTATION = 4;   // good default value
    static const uint32_t SHM_LIMIT = 0;      // no shared mem available
    static const bool CONSTANT_TIME = false;  // constant time implementations aren't available yet

    // parameters used locally in the application
    static const uint32_t TPI = tpi;    // threads per instance
    static const uint32_t BITS = bits;  // instance size
};

template <class params>
class gsv_t {
   public:
    // instance_t should be 128-byte aligned
    typedef struct {
        cgbn_mem_t<params::BITS> r;      // sig->r
        cgbn_mem_t<params::BITS> s;      // sig->s
        cgbn_mem_t<params::BITS> e;      // digest
        cgbn_mem_t<params::BITS> key_x;  // public key
        cgbn_mem_t<params::BITS> key_y;  // public key
    } instance_t;

    typedef struct {
        cgbn_mem_t<params::BITS> order;  // group order
        cgbn_mem_t<params::BITS> g_x;    // base point (generator)
        cgbn_mem_t<params::BITS> g_y;    // base point (generator)
        cgbn_mem_t<params::BITS> field;  // prime p
        cgbn_mem_t<params::BITS> g_a;
    } ec_t;

    typedef cgbn_context_t<params::TPI> context_t;
    typedef cgbn_env_t<context_t, params::BITS> env_t;
    typedef typename env_t::cgbn_t bn_t;
    typedef typename env_t::cgbn_local_t bn_local_t;
    typedef typename env_t::cgbn_wide_t bn_wide_t;

    context_t _context;
    env_t _env;
    int32_t _instance;

    __device__ __forceinline__ gsv_t(cgbn_monitor_t monitor, cgbn_error_report_t *report, int32_t instance)
        : _context(monitor, report, (uint32_t)instance), _env(_context), _instance(instance) {}

    __device__ __forceinline__ void mod(bn_t &r, const bn_t &m) {
        while (_env.compare(r, m) >= 0) {
            _env.sub(r, r, m);
        }
    }

    // fast modular addition: r = (a + b) mod m
    // both a and b should be non-negative and less than m
    __device__ __forceinline__ void mod_add(bn_t &r, const bn_t &a, const bn_t &b, const bn_t &m) {
#ifdef BIT256
        if (_env.add(r, a, b) || _env.compare(r, m) >= 0) {
            _env.sub(r, r, m);
        }
#else
        _env.add(r, a, b);
        if (_env.compare(r, m) >= 0) {
            _env.sub(r, r, m);
        }
#endif
    }

    // r = (a - b) mod m
    __device__ __forceinline__ void mod_sub(bn_t &r, const bn_t &a, const bn_t &b, const bn_t &m) {
        if (_env.sub(r, a, b)) {  // a < b
            _env.add(r, r, m);
        }
    }

    // r = (a * 2) mod m
    __device__ __forceinline__ void mod_lshift1(bn_t &r, const bn_t &a, const bn_t &m) {
#ifdef BIT256
        uint32_t z = _env.clz(a);
        _env.shift_left(r, a, 1);
        if (z == 0 || _env.compare(r, m) >= 0) {
            _env.sub(r, r, m);
        }
#else
        _env.shift_left(r, a, 1);
        if (_env.compare(r, m) >= 0) {
            _env.sub(r, r, m);
        }
#endif
    }

    // not used
    __device__ __forceinline__ void mod_lshift(bn_t &r, const bn_t &a, const bn_t &m, uint32_t n) {
        for (uint32_t i = 0; i < n; i++) {
            mod_lshift1(r, a, m);
        }
    }

    // OpenSSL's point doubling. Buggy, do not use
    // Complexity: 6S, 4M, 2A, 3D, 3L, 1L2, 1L3
    __device__ __forceinline__ void point_dbl(bn_t &r_x, bn_t &r_y, bn_t &r_z, const bn_t &a_x, const bn_t &a_y,
                                              const bn_t &a_z, const bn_t &field, const bn_t &g_a, const uint32_t np0) {
        if (_env.equals_ui32(a_z, 0)) {
            _env.set_ui32(r_z, 0);
            return;
        }

        bn_t n0, n1, n2, n3;

        _env.mont_sqr(n0, a_x, field, np0);      // n0 = a_x^2
        mod_lshift1(n1, n0, field);              // n1 = 2 * a_x^2
        mod_add(n0, n0, n1, field);              // n0 = 3 * a_x^2
        _env.mont_sqr(n1, a_z, field, np0);      // n1 = a_z^2
        _env.mont_sqr(n1, n1, field, np0);       // n1 = a_z^4
        _env.mont_mul(n1, n1, g_a, field, np0);  // n1 = g_a * a_z^4
        mod_add(n1, n1, n0, field);              // n1 = 3 * a_x^2 + g_a * a_z^4

        _env.mont_mul(n0, a_y, a_z, field, np0);  // n0 = a_y * a_z
        mod_lshift1(r_z, n0, field);              // r_z = 2 * a_y * a_z

        _env.mont_sqr(n3, a_y, field, np0);      // n3 = a_y^2
        _env.mont_mul(n2, a_x, n3, field, np0);  // n2 = a_x * a_y^2
        mod_lshift(n2, n2, field, 2);            // n2 = 4 * a_x * a_y^2

        mod_lshift1(n0, n2, field);          // n0 = 2 * n2
        _env.mont_sqr(r_x, n1, field, np0);  // r_x = n1^2
        mod_sub(r_x, r_x, n0, field);        // r_x = n1^2 - 2 * n2

        _env.mont_sqr(n0, n3, field, np0);  // n0 = a_y^4
        mod_lshift(n3, n0, field, 3);       // n3 = 8 * a_y^4

        mod_sub(n0, n2, r_x, field);            // n0 = n2 - r_x
        _env.mont_mul(n0, n1, n0, field, np0);  // n0 = n1 * (n2 - r_x)
        mod_sub(r_y, n0, n3, field);            // r_y = n1 * (n2 - r_x) - n3
    }

    // Intel IPP's faster point doubling
    // Complexity: 6S, 4M, 2A, 3D, 3L, 1R
    // SM2:        4S, 4M, 2A, 4D, 3L, 1R
    __device__ __forceinline__ void point_dbl_ipp(bn_t &r_x, bn_t &r_y, bn_t &r_z, const bn_t &a_x, const bn_t &a_y,
                                                  const bn_t &a_z, const bn_t &field, const bn_t &g_a, const uint32_t np0) {
        if (_env.equals_ui32(a_z, 0)) {
            _env.set_ui32(r_z, 0);
            return;
        }

        bn_t u, m, s, t;

        mod_lshift1(s, a_y, field);         // s = 2 * a_y
        _env.mont_sqr(u, a_z, field, np0);  // u = a_z^2

        _env.mont_sqr(m, s, field, np0);         // m = 4 * a_y^2
        _env.mont_mul(r_z, s, a_z, field, np0);  // r_z = 2 * a_y * a_z

        _env.mont_sqr(r_y, m, field, np0);  // r_y = 16 * a_y^4

        _env.mont_mul(s, m, a_x, field, np0);  // s = 4 * a_x * a_y^2

#ifdef BIT256
        if (_env.ctz(r_y) == 0 && _env.add(r_y, r_y, field)) {
            _env.shift_right(r_y, r_y, 1);
            _env.bitwise_mask_ior(r_y, r_y, -1);
        } else {
            _env.shift_right(r_y, r_y, 1);  // r_y = 8 * a_y^4
        }
#else
        if (_env.ctz(r_y) == 0) {
            _env.add(r_y, r_y, field);
        }
        _env.shift_right(r_y, r_y, 1);      // r_y = 8 * a_y^4
#endif

#ifdef SM2
        mod_add(m, a_x, u, field);           // m = a_x + u
        mod_sub(u, a_x, u, field);           // u = a_x - u
        _env.mont_mul(m, m, u, field, np0);  // m = (a_x + u) * (a_x - u) = a_x^2 - a_z^4
        mod_lshift1(t, m, field);            // t = 2 * (a_x^2 - a_z^4)
        mod_add(m, m, t, field);             // m = 3 * (a_x^2 - a_z^4)
#else
        _env.mont_sqr(m, a_x, field, np0);  // m = a_x ^ 2
        mod_lshift1(t, m, field);           // t = 2 * a_x^2
        mod_add(m, m, t, field);            // m = 3 * a_x^2

        _env.mont_sqr(u, u, field, np0);       // u = a_z^4
        _env.mont_mul(u, u, g_a, field, np0);  // u = g_a * a_z^4
        mod_add(m, m, u, field);               // m = 3 * a_x^2 + g_a * a_z^4
#endif

        mod_lshift1(u, s, field);           // u = 8 * a_x * a_y^2
        _env.mont_sqr(r_x, m, field, np0);  // r_x = m^2
        mod_sub(r_x, r_x, u, field);        // r_x = m^2 - u

        mod_sub(s, s, r_x, field);           // s = 4 * a_x * a_y^2 - r_x
        _env.mont_mul(s, s, m, field, np0);  // s = (4 * a_x * a_y^2 - r_x) * m
        mod_sub(r_y, s, r_y, field);         // r_y = (4 * a_x * a_y^2 - r_x) * m - 8 * a_y^4
    }

    // OpenSSL's point addition
    // Complexity: 4S, 12M, 2A, 5D, 1L, 1R
    __device__ __forceinline__ void point_add(bn_t &r_x, bn_t &r_y, bn_t &r_z, const bn_t &a_x, const bn_t &a_y,
                                              const bn_t &a_z, const bn_t &b_x, const bn_t &b_y, const bn_t &b_z,
                                              const bn_t &field, const bn_t &g_a, const uint32_t np0) {
        if (_env.compare(a_x, b_x) == 0 && _env.compare(a_y, b_y) == 0 && _env.compare(a_z, b_z) == 0) {
            // if (threadIdx.x == 0) printf("DOUBLE\n");
            point_dbl(r_x, r_y, r_z, a_x, a_y, a_z, field, g_a, np0);
            return;
        }
        if (_env.equals_ui32(a_z, 0)) {
            _env.set(r_x, b_x);
            _env.set(r_y, b_y);
            _env.set(r_z, b_z);
            return;
        }
        if (_env.equals_ui32(b_z, 0)) {
            _env.set(r_x, a_x);
            _env.set(r_y, a_y);
            _env.set(r_z, a_z);
            return;
        }

        bn_t n0, n1, n2, n3, n4, n5, n6;

        _env.mont_sqr(n0, b_z, field, np0);      // n0 = b_z^2
        _env.mont_mul(n1, a_x, n0, field, np0);  // n1 = a_x * b_z^2

        _env.mont_mul(n0, n0, b_z, field, np0);  // n0 = b_z^3
        _env.mont_mul(n2, a_y, n0, field, np0);  // n2 = a_y * b_z^3

        _env.mont_sqr(n0, a_z, field, np0);      // n0 = a_z^2
        _env.mont_mul(n3, b_x, n0, field, np0);  // n3 = b_x * a_z^2

        _env.mont_mul(n0, n0, a_z, field, np0);  // n0 = a_z^3
        _env.mont_mul(n4, b_y, n0, field, np0);  // n4 = b_y * a_z^3

        mod_sub(n5, n1, n3, field);  // n5 = n1 - n3
        mod_sub(n6, n2, n4, field);  // n6 = n2 - n4

        if (_env.equals_ui32(n5, 0)) {
            if (_env.equals_ui32(n6, 0)) {
                point_dbl(r_x, r_y, r_z, a_x, a_y, a_z, field, g_a, np0);
                return;
            } else {
                _env.set_ui32(r_z, 0);
                return;
            }
        }

        mod_add(n1, n1, n3, field);  // 'n7' = n1 + n3
        mod_add(n2, n2, n4, field);  // 'n8' = n2 + n4

        _env.mont_mul(n0, a_z, b_z, field, np0);  // n0 = a_z * b_z
        _env.mont_mul(r_z, n0, n5, field, np0);   // r_z = a_z * b_z * n5

        _env.mont_sqr(n0, n6, field, np0);      // n0 = n6^2
        _env.mont_sqr(n4, n5, field, np0);      // n4 = n5^2
        _env.mont_mul(n3, n1, n4, field, np0);  // n3 = n5^2 * 'n7'
        mod_sub(r_x, n0, n3, field);            // r_x = n6^2 - n5^2 * 'n7'

        mod_lshift1(n0, r_x, field);  // n0 = 2 * r_x
        mod_sub(n0, n3, n0, field);   // 'n9' = n5^2 * 'n7' - 2 * r_x

        _env.mont_mul(n0, n0, n6, field, np0);  // n0 = n6 * 'n9'
        _env.mont_mul(n5, n4, n5, field, np0);  // 'n5' = n5^3
        _env.mont_mul(n1, n2, n5, field, np0);  // n1 = 'n8' * n5^3
        mod_sub(n0, n0, n1, field);             // n0 = n6 * 'n9' - 'n8' * n5^3
        if (_env.ctz(n0) == 0) {                // if n0 is odd
            _env.add(n0, n0, field);            // 0 <= n0 < 2 * field, n0 is even
        }
        _env.shift_right(r_y, n0, 1);  // r_y = (n6 * 'n9' - 'n8' * n5^3) / 2
    }

    // Intel IPP's faster point addition
    // Complexity: 4S, 12M, 0A, 6D, 1L
    __device__ __forceinline__ void point_add_ipp(bn_t &r_x, bn_t &r_y, bn_t &r_z, const bn_t &a_x, const bn_t &a_y,
                                                  const bn_t &a_z, const bn_t &b_x, const bn_t &b_y, const bn_t &b_z,
                                                  const bn_t &field, const bn_t &g_a, const uint32_t np0) {
        if (_env.compare(a_x, b_x) == 0 && _env.compare(a_y, b_y) == 0 && _env.compare(a_z, b_z) == 0) {
            // if (threadIdx.x == 0) printf("DOUBLE\n");
            point_dbl_ipp(r_x, r_y, r_z, a_x, a_y, a_z, field, g_a, np0);
            return;
        }
        if (_env.equals_ui32(a_z, 0)) {
            _env.set(r_x, b_x);
            _env.set(r_y, b_y);
            _env.set(r_z, b_z);
            return;
        }
        if (_env.equals_ui32(b_z, 0)) {
            _env.set(r_x, a_x);
            _env.set(r_y, a_y);
            _env.set(r_z, a_z);
            return;
        }

        bn_t u1, u2, s1, s2, h, r;

        _env.mont_mul(s1, a_y, b_z, field, np0);  // s1 = a_y * b_z
        _env.mont_sqr(u1, b_z, field, np0);       // u1 = b_z^2

        _env.mont_mul(s2, b_y, a_z, field, np0);  // s2 = b_y * a_z
        _env.mont_sqr(u2, a_z, field, np0);       // u2 = a_z^2

        _env.mont_mul(s1, s1, u1, field, np0);  // s1 = a_y * b_z^3
        _env.mont_mul(s2, s2, u2, field, np0);  // s2 = b_y * a_z^3

        _env.mont_mul(u1, a_x, u1, field, np0);  // u1 = a_x * b_z^2
        _env.mont_mul(u2, b_x, u2, field, np0);  // u2 = b_x * a_z^2

        mod_sub(r, s2, s1, field);  // r = s2 - s1
        mod_sub(h, u2, u1, field);  // h = u2 - u1

        if (_env.equals_ui32(h, 0)) {
            if (_env.equals_ui32(r, 0)) {
                // if (threadIdx.x == 0) printf("EQUAL\n");
                point_dbl_ipp(r_x, r_y, r_z, a_x, a_y, a_z, field, g_a, np0);
                return;
            } else {
                _env.set_ui32(r_z, 0);
                return;
            }
        }

        _env.mont_mul(r_z, a_z, b_z, field, np0);  // r_z = a_z * b_z
        _env.mont_sqr(u2, h, field, np0);          // u2 = h^2
        _env.mont_mul(r_z, r_z, h, field, np0);    // r_z = a_z * b_z * h
        _env.mont_sqr(s2, r, field, np0);          // s2 = r^2
        _env.mont_mul(h, h, u2, field, np0);       // h = h^3

        _env.mont_mul(u1, u1, u2, field, np0);  // u1 = u1 * h^2
        mod_sub(r_x, s2, h, field);             // r_x = r^2 - h^3
        mod_lshift1(u2, u1, field);             // u2 = 2 * u1 * h^2
        _env.mont_mul(s1, s1, h, field, np0);   // s1 = s1 * h^3
        mod_sub(r_x, r_x, u2, field);           // r_x = r^2 - h^3 - 2 * u1 * h^2

        mod_sub(r_y, u1, r_x, field);            // r_y = u1 * h^2 - r_x
        _env.mont_mul(r_y, r_y, r, field, np0);  // r_y = r * (u1 * h^2 - r_x)
        mod_sub(r_y, r_y, s1, field);            // r_y = r * (u1 * h^2 - r_x) - s1 * h^3
    }

    // double-and-add, index increasing
    // Expected complexity: n * D + n/2 * A
    __device__ __forceinline__ void point_mult(bn_t &r_x, bn_t &r_y, bn_t &r_z, const bn_t &p_x, const bn_t &p_y,
                                               const bn_t &p_z, const bn_t &d, const bn_t &field, const bn_t &g_a,
                                               const uint32_t np0) {
        bn_t q_x, q_y, q_z;
        bn_t k;

        _env.set(k, d);
        _env.set(q_x, p_x);
        _env.set(q_y, p_y);
        _env.set(q_z, p_z);
        _env.set_ui32(r_z, 0);

        while (_env.compare_ui32(k, 0) > 0) {
            if (_env.ctz(k) == 0) {  // k_i = 1
                point_add_ipp(r_x, r_y, r_z, r_x, r_y, r_z, q_x, q_y, q_z, field, g_a, np0);
            }
            point_dbl_ipp(q_x, q_y, q_z, q_x, q_y, q_z, field, g_a, np0);
            _env.shift_right(k, k, 1);
        }
    }

    // double-and-add, use shared memory to store d
    __device__ __forceinline__ void point_mult_shared(bn_t &r_x, bn_t &r_y, bn_t &r_z, const bn_t &p_x, const bn_t &p_y,
                                                      const bn_t &p_z, const bn_t &d, const bn_t &field, const bn_t &g_a,
                                                      const uint32_t np0) {
        bn_t q_x, q_y, q_z;
        uint32_t limb;
        __shared__ cgbn_mem_t<params::BITS> s_d;

        _env.store(&s_d, d);
        _env.set(q_x, p_x);
        _env.set(q_y, p_y);
        _env.set(q_z, p_z);
        _env.set_ui32(r_z, 0);

        for (int i = 0; i < 8; i++) {  // 256-bit integer
            limb = s_d._limbs[i];
            // if (limb == 0) {  // this useless 'if' can improve 256/512 instances performance...
            //     break;
            // }
            for (int j = 0; j < 32; j++) {
                if (limb & 1) {
                    // if (threadIdx.x == 0) printf("%d\t%d:\t%d\t%d\t%u\t%u\n", blockIdx.x, threadIdx.x, i, j, limb, mask);
                    point_add_ipp(r_x, r_y, r_z, r_x, r_y, r_z, q_x, q_y, q_z, field, g_a, np0);
                }
                point_dbl_ipp(q_x, q_y, q_z, q_x, q_y, q_z, field, g_a, np0);
                limb >>= 1;
            }
        }
    }

    // double-and-add, index decreasing
    __device__ __forceinline__ void point_mult_desc(bn_t &r_x, bn_t &r_y, bn_t &r_z, const bn_t &p_x, const bn_t &p_y,
                                                    const bn_t &p_z, const bn_t &d, const bn_t &field, const bn_t &g_a,
                                                    const uint32_t np0) {
        bn_t q_x, q_y, q_z;
        uint32_t limb;
        __shared__ cgbn_mem_t<params::BITS> s_d;

        _env.store(&s_d, d);
        _env.set(q_x, p_x);
        _env.set(q_y, p_y);
        _env.set(q_z, p_z);
        _env.set_ui32(r_z, 0);

        // int bits = (params::BITS + 31) / 32;
        int flag = 0;
        for (int i = 7; i >= 0; i--) {
            limb = s_d._limbs[i];
            // if (limb == 0) {
            //     continue;
            // }
            uint32_t mask = 0x80000000L;
            for (int j = 0; j < 32; j++) {
                if ((!flag) && (limb & mask)) {
                    flag = 1;
                }
                if (flag) {
                    point_dbl_ipp(r_x, r_y, r_z, r_x, r_y, r_z, field, g_a, np0);
                }
                if (limb & mask) {
                    // if (threadIdx.x == 0) printf("%d\t%d:\t%d\t%d\t%u\t%u\n", blockIdx.x, threadIdx.x, i, j, limb, mask);
                    point_add_ipp(r_x, r_y, r_z, r_x, r_y, r_z, q_x, q_y, q_z, field, g_a, np0);
                }
                mask >>= 1;
            }
        }
    }

    // Non-adjacent form (NAF)
    // Expected complexity: n * D + n/3 * A
    __device__ __forceinline__ void point_mult_naf(bn_t &r_x, bn_t &r_y, bn_t &r_z, const bn_t &p_x, const bn_t &p_y,
                                                   const bn_t &p_z, const bn_t &d, const bn_t &field, const bn_t &g_a,
                                                   const uint32_t np0) {
        bn_t q_x, q_y, q_z;
        bn_t k, m_y;
        int8_t naf[257];

        _env.set(q_x, p_x);
        _env.set(q_y, p_y);
        _env.set(q_z, p_z);
        _env.set(k, d);
        _env.set_ui32(r_z, 0);

        _env.sub(m_y, field, p_y);  // my = -p_y mod field

        int bits = 0;
        while (_env.compare_ui32(k, 0) > 0) {
            if (_env.ctz(k) == 0) {  // k is odd
                _env.shift_right(k, k, 1);
                if (_env.ctz(k) == 0) {  // k mod 4 = 3
                    naf[bits] = -1;
                    _env.add_ui32(k, k, 1);
                } else {  // k mod 4 = 1;
                    naf[bits] = 1;
                }
            } else {
                _env.shift_right(k, k, 1);
                naf[bits] = 0;
            }
            ++bits;
        }

        for (int i = bits - 1; i >= 0; i--) {
            point_dbl_ipp(r_x, r_y, r_z, r_x, r_y, r_z, field, g_a, np0);
            if (naf[i] == 1) {
                point_add_ipp(r_x, r_y, r_z, r_x, r_y, r_z, q_x, q_y, q_z, field, g_a, np0);
            } else if (naf[i] == -1) {
                point_add_ipp(r_x, r_y, r_z, r_x, r_y, r_z, q_x, m_y, q_z, field, g_a, np0);
            }
        }
    }

    // wNAF: width-w NAF. needs to pre-compute iP for i={1,3,5,...,2^{w-1}-1}.
    // Expected complexity: 1 * D + (2^{w-2}-1) * A (pre-computation), n * D + n/(w+1) * A
    __device__ __forceinline__ void point_mult_wnaf(bn_t &r_x, bn_t &r_y, bn_t &r_z, const bn_t &p_x, const bn_t &p_y,
                                                    const bn_t &p_z, const bn_t &d, const bn_t &field, const bn_t &g_a,
                                                    const uint32_t np0) {}

    // transform (X, Y, Z) into (x, y) := (X/Z^2, Y/Z^3)
    __device__ __forceinline__ void conv_affine_x_y(bn_t &a_x, bn_t &a_y, const bn_t &j_x, const bn_t &j_y, const bn_t &j_z,
                                                    const bn_t &field, const uint32_t np0) {
        if (_env.equals_ui32(j_z, 0)) {
            _env.set_ui32(a_x, 1);
            _env.set_ui32(a_y, 1);
            return;
        }

        bn_t Z_, Z_1, Z_2, Z_3;

        _env.mont2bn(Z_, j_z, field, np0);

        if (_env.equals_ui32(Z_, 1)) {
            _env.mont2bn(a_x, j_x, field, np0);
            _env.mont2bn(a_y, j_y, field, np0);
        } else {
            _env.modular_inverse(Z_1, Z_, field);
            _env.bn2mont(Z_1, Z_1, field);
            _env.mont_sqr(Z_2, Z_1, field, np0);
            _env.mont_mul(a_x, j_x, Z_2, field, np0);
            _env.mont2bn(a_x, a_x, field, np0);
        }
    }

    // transform (X, Y, Z) into x := X/Z^2
    __device__ __forceinline__ void conv_affine_x(bn_t &a_x, const bn_t &j_x, const bn_t &j_z, const bn_t &field,
                                                  const uint32_t np0) {
        if (_env.equals_ui32(j_z, 0)) {
            _env.set_ui32(a_x, 1);
            return;
        }

        bn_t Z_, Z_1, Z_2;

        _env.mont2bn(Z_, j_z, field, np0);

        if (_env.equals_ui32(Z_, 1)) {
            _env.mont2bn(a_x, j_x, field, np0);
        } else {
            _env.modular_inverse(Z_1, Z_, field);
            _env.bn2mont(Z_1, Z_1, field);
            _env.mont_sqr(Z_2, Z_1, field, np0);
            _env.mont_mul(a_x, j_x, Z_2, field, np0);
            _env.mont2bn(a_x, a_x, field, np0);
        }

        // _env.modular_inverse(Z_1, j_z, field);
    }

#ifdef DEBUG
    __device__ __forceinline__ int32_t debug_kernel(const bn_t &r, const bn_t &s, const bn_t &e, const bn_t &key_x,
                                                    const bn_t &key_y, const bn_t &order, const bn_t &g_x, const bn_t &g_y,
                                                    const bn_t &field, bn_t &g_a, bn_t &tmp) {
        bn_t x1, y1, z1, x2, y2, one, zero;
        uint32_t np0;

        _env.set_ui32(zero, 0);
        _env.set_ui32(one, 1);
        np0 = _env.bn2mont(one, one, field);
        mod(g_a, field);
        _env.bn2mont(g_a, g_a, field);

        _env.set(x1, g_x);
        _env.set(y1, g_y);
        mod(x1, field);
        _env.bn2mont(x1, x1, field);
        mod(y1, field);
        _env.bn2mont(y1, y1, field);

        _env.set(x2, key_x);
        _env.set(y2, key_y);
        mod(x2, field);
        _env.bn2mont(x2, x2, field);
        mod(y2, field);
        _env.bn2mont(y2, y2, field);

        // point_add(x1, y1, z1, x1, y1, one, x2, y2, one, field, g_a, np0);
        point_add(x1, y1, z1, x2, y2, one, x1, y1, one, field, g_a, np0);
        // point_add_ipp(x1, y1, z1, x1, y1, one, x2, y2, one, field, g_a, np0);
        // point_add_ipp(x1, y1, z1, x2, y2, one, x1, y1, one, field, g_a, np0);
        // point_add(x1, y1, z1, x1, y1, one, one, one, zero, field, g_a, np0);
        // point_add(x1, y1, z1, one, one, zero, x1, y1, one, field, g_a, np0);

        // _env.set(tmp, z1);
        conv_affine_x(tmp, x1, z1, field, np0);

        // point_add(x1, y1, z1, x1, y1, z1, r, s, one, field, g_a, np0);
        // point_add_ipp(x1, y1, z1, x1, y1, z1, r, s, one, field, g_a, np0);

        return 0;
    }
#endif

    /*
     * B1: verify whether r' in [1,n-1], verification failed if not
     * B2: verify whether s' in [1,n-1], verification failed if not
     * B3: set M'~=ZA || M'
     * B4: calculate e'=Hv(M'~)
     * B5: calculate t = (r' + s') modn, verification failed if t=0
     * B6: calculate the point (x1', y1')=[s']G + [t]PA
     * B7: calculate R=(e'+x1') modn, verification pass if yes, otherwise failed
     */
#ifdef DEBUG
    __device__ __forceinline__ int32_t sig_verify(const bn_t &r, bn_t &s, const bn_t &e, const bn_t &key_x, const bn_t &key_y,
                                                  const bn_t &order, const bn_t &g_x, const bn_t &g_y, const bn_t &field,
                                                  bn_t &g_a, bn_t &tmp)
#else
    __device__ __forceinline__ int32_t sig_verify(const bn_t &r, const bn_t &s, const bn_t &e, const bn_t &key_x,
                                                  const bn_t &key_y, const bn_t &order, const bn_t &g_x, const bn_t &g_y,
                                                  const bn_t &field, bn_t &g_a)
#endif
    {
        bn_t t, x1, y1, z1, x2, y2, z2;
        uint32_t np0;

        if (_env.compare_ui32(r, 1) < 0 || _env.compare_ui32(s, 1) < 0 || _env.compare(order, r) <= 0 ||
            _env.compare(order, s) <= 0) {
            return 0;
        }

        mod_add(t, r, s, order);

        if (_env.equals_ui32(t, 0)) {
            return 0;
        }

        _env.set_ui32(z1, 1);
        np0 = _env.bn2mont(z1, z1, field);
        _env.set(z2, z1);

        mod(g_a, field);
        _env.bn2mont(g_a, g_a, field);

        // s * generator + t * pkey
        _env.set(x1, g_x);
        _env.set(y1, g_y);
        mod(x1, field);
        _env.bn2mont(x1, x1, field);
        mod(y1, field);
        _env.bn2mont(y1, y1, field);
        point_mult_naf(x1, y1, z1, x1, y1, z1, s, field, g_a, np0);

        __syncthreads();  // TODO: temp fix of wrong answer, need to test on different input

        _env.set(x2, key_x);
        _env.set(y2, key_y);
        mod(x2, field);
        _env.bn2mont(x2, x2, field);
        mod(y2, field);
        _env.bn2mont(y2, y2, field);
        point_mult_naf(x2, y2, z2, x2, y2, z2, t, field, g_a, np0);

        point_add(x1, y1, z1, x1, y1, z1, x2, y2, z2, field, g_a, np0);

        conv_affine_x(x1, x1, z1, field, np0);

        mod_add(t, e, x1, order);

        return _env.compare(r, t);
    }

    __host__ static instance_t *generate_instances(uint32_t count) {
        instance_t *instances = (instance_t *)malloc(sizeof(instance_t) * count);

        for (int index = 0; index < count; index++) {
#ifdef SM2
            set_words(instances[index].r._limbs, "23B20B796AAAFEAAA3F1592CB9B4A93D5A8D279843E1C57980E64E0ABC5F5B05",
                      params::BITS / 32);
            set_words(instances[index].s._limbs, "E11F5909F947D5BE08C84A22CE9F7C338F7CF4A5B941B9268025495D7D433071",
                      params::BITS / 32);
            set_words(instances[index].key_x._limbs, "D5548C7825CBB56150A3506CD57464AF8A1AE0519DFAF3C58221DC810CAF28DD",
                      params::BITS / 32);
            set_words(instances[index].key_y._limbs, "921073768FE3D59CE54E79A49445CF73FED23086537027264D168946D479533E",
                      params::BITS / 32);
            set_words(instances[index].e._limbs, "10D51CB90C0C0522E94875A2BEA7AB72299EBE7192E64EFE0573B1C77110E5C9",
                      params::BITS / 32);
#else
            // #ifdef DEBUG
            //       set_words(instances[index].r._limbs, "40F1EC59F793D9F49E09DCEF49130D4194F79FB1EED2CAA55BACDB49C4E755D1",
            //                 params::BITS / 32);
            //       set_words(instances[index].s._limbs, "6FC6DAC32C5D5CF10C77DFB20F7C2EB667A457872FB09EC56327A67EC7DEEBE7",
            //                 params::BITS / 32);
            //       set_words(instances[index].key_x._limbs,
            //       "7DEACE5FD121BC385A3C6317249F413D28C17291A60DFD83B835A45392D22B0A",
            //                 params::BITS / 32);
            //       set_words(instances[index].key_y._limbs,
            //       "2E49D5E5279E5FA91E71FD8F693A64A3C4A9461115A4FC9D79F34EDC8BDDEBD0",
            //                 params::BITS / 32);
            // #else
            set_words(instances[index].r._limbs, "40F1EC59F793D9F49E09DCEF49130D4194F79FB1EED2CAA55BACDB49C4E755D1",
                      params::BITS / 32);
            set_words(instances[index].s._limbs, "6FC6DAC32C5D5CF10C77DFB20F7C2EB667A457872FB09EC56327A67EC7DEEBE7",
                      params::BITS / 32);
            set_words(instances[index].key_x._limbs, "AE4C7798AA0F119471BEE11825BE46202BB79E2A5844495E97C04FF4DF2548A",
                      params::BITS / 32);
            set_words(instances[index].key_y._limbs, "7C0240F88F1CD4E16352A73C17B7F16F07353E53A176D684A9FE0C6BB798E857",
                      params::BITS / 32);
            // #endif
            set_words(instances[index].e._limbs, "B524F552CD82B8B028476E005C377FB19A87E6FC682D48BB5D42E3D9B9EFFE76",
                      params::BITS / 32);
#endif
        }
        return instances;
    }

    __host__ static void verify_results(instance_t *instances, uint32_t count, int32_t *results) {
        for (int index = 0; index < count; index++) {
            int openssl_result = -1;

            // TODO: call OpenSSL sig verify here for cross validation
            openssl_result = 0;

#ifdef DEBUG
            print_words(instances[index].r._limbs, params::BITS / 32);
#endif

            if (openssl_result != results[index]) {
                printf("Wrong result %d on instance %d\n", results[index], index);
                break;
            }
        }
    }
};

template <class params>
__global__ void kernel_sig_verify(cgbn_error_report_t *report, typename gsv_t<params>::instance_t *instances,
                                  uint32_t instance_count, typename gsv_t<params>::ec_t ec, int32_t *results) {
    int32_t instance = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
    if (instance >= instance_count) return;

    typedef gsv_t<params> local_gsv_t;

    local_gsv_t gsv(cgbn_report_monitor, report, instance);
    typename local_gsv_t::bn_t r, s, e, key_x, key_y, order, g_x, g_y, field, g_a;

#ifdef DEBUG
    typename local_gsv_t::bn_t tmp;
#endif

    cgbn_load(gsv._env, r, &(instances[instance].r));
    cgbn_load(gsv._env, s, &(instances[instance].s));
    cgbn_load(gsv._env, e, &(instances[instance].e));
    cgbn_load(gsv._env, key_x, &(instances[instance].key_x));
    cgbn_load(gsv._env, key_y, &(instances[instance].key_y));

    cgbn_load(gsv._env, order, &(ec.order));
    cgbn_load(gsv._env, g_x, &(ec.g_x));
    cgbn_load(gsv._env, g_y, &(ec.g_y));
    cgbn_load(gsv._env, field, &(ec.field));
    cgbn_load(gsv._env, g_a, &(ec.g_a));

#ifdef DEBUG
    results[instance] = gsv.sig_verify(r, s, e, key_x, key_y, order, g_x, g_y, field, g_a, tmp);
    // results[instance] = gsv.debug_kernel(r, s, e, key_x, key_y, order, g_x, g_y, field, g_a, tmp);
    cgbn_store(gsv._env, &(instances[instance].r), tmp);
#else
    results[instance] = gsv.sig_verify(r, s, e, key_x, key_y, order, g_x, g_y, field, g_a);
#endif
}

template <class params>
void test_sig_verify(uint32_t instance_count, typename gsv_t<params>::instance_t *d_instances, int32_t *d_results,
                     cgbn_error_report_t *report) {
    typedef typename gsv_t<params>::instance_t instance_t;
    typedef typename gsv_t<params>::ec_t ec_t;

    instance_t *instances;
    ec_t sm2;
    int32_t *results;                                      // signature verification result, 0 is true, 1 is false
    int32_t TPB = (params::TPB == 0) ? 128 : params::TPB;  // default threads per block is 128
    int32_t TPI = params::TPI, IPB = TPB / TPI;            // IPB: instances per block

    results = (int32_t *)malloc(sizeof(int32_t) * instance_count);
    instances = gsv_t<params>::generate_instances(instance_count);

#ifdef SM2
    set_words(sm2.order._limbs, "FFFFFFFEFFFFFFFFFFFFFFFFFFFFFFFF7203DF6B21C6052B53BBF40939D54123", params::BITS / 32);
    set_words(sm2.g_x._limbs, "32C4AE2C1F1981195F9904466A39C9948FE30BBFF2660BE1715A4589334C74C7", params::BITS / 32);
    set_words(sm2.g_y._limbs, "BC3736A2F4F6779C59BDCEE36B692153D0A9877CC62A474002DF32E52139F0A0", params::BITS / 32);
    set_words(sm2.field._limbs, "FFFFFFFEFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFF00000000FFFFFFFFFFFFFFFF", params::BITS / 32);
    set_words(sm2.g_a._limbs, "FFFFFFFEFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFF00000000FFFFFFFFFFFFFFFC", params::BITS / 32);
#else
    set_words(sm2.order._limbs, "8542D69E4C044F18E8B92435BF6FF7DD297720630485628D5AE74EE7C32E79B7", params::BITS / 32);
    // #ifdef DEBUG
    //   set_words(sm2.g_x._limbs, "1657FA75BF2ADCDC3C1F6CF05AB7B45E04D3ACBE8E4085CFA669CB2564F17A9F", params::BITS / 32);
    //   set_words(sm2.g_y._limbs, "19F0115F21E16D2F5C3A485F8575A128BBCDDF80296A62F6AC2EB842DD058E50", params::BITS / 32);
    // #else
    set_words(sm2.g_x._limbs, "421DEBD61B62EAB6746434EBC3CC315E32220B3BADD50BDC4C4E6C147FEDD43D", params::BITS / 32);
    set_words(sm2.g_y._limbs, "0680512BCBB42C07D47349D2153B70C4E5D7FDFCBFA36EA1A85841B9E46E09A2", params::BITS / 32);
    // #endif
    set_words(sm2.field._limbs, "8542D69E4C044F18E8B92435BF6FF7DE457283915C45517D722EDB8B08F1DFC3", params::BITS / 32);
    set_words(sm2.g_a._limbs, "787968B4FA32C3FD2417842E73BBFEFF2F3C848B6831D7E0EC65228B3937E498", params::BITS / 32);
#endif

    auto t_start = std::chrono::high_resolution_clock::now();

    CUDA_CHECK(hipMemcpy(d_instances, instances, sizeof(instance_t) * instance_count, hipMemcpyHostToDevice));

    auto k_start = std::chrono::high_resolution_clock::now();

    kernel_sig_verify<params><<<(instance_count + IPB - 1) / IPB, TPB>>>(report, d_instances, instance_count, sm2, d_results);

    CUDA_CHECK(hipDeviceSynchronize());
    CGBN_CHECK(report);

    auto k_end = std::chrono::high_resolution_clock::now();

#ifdef DEBUG
    CUDA_CHECK(hipMemcpy(instances, d_instances, sizeof(instance_t) * instance_count, hipMemcpyDeviceToHost));
#endif

    CUDA_CHECK(hipMemcpy(results, d_results, sizeof(int32_t) * instance_count, hipMemcpyDeviceToHost));

    auto t_end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> t_diff = t_end - t_start;
    std::chrono::duration<double> k_diff = k_end - k_start;

    printf("Wall time: %lfs (Mem transfer %lfs), Speed: %lfV/s (w/o mem transfer: %lfV/s)\n", t_diff.count(),
           t_diff.count() - k_diff.count(), (double)instance_count / t_diff.count(), (double)instance_count / k_diff.count());

    gsv_t<params>::verify_results(instances, instance_count, results);

    free(instances);
    free(results);
}

#define MAX_INS 262144

int main(int argc, char **argv) {
    int device_id = 0;
    if (argc >= 2) {
        device_id = atoi(argv[1]);
    }
    CUDA_CHECK(hipSetDevice(device_id));

#ifdef BIT256
    typedef gsv_params_t<16, 256> params;  // threads per instance, instance size
#else
    typedef gsv_params_t<16, 512> params;  // threads per instance, instance size
#endif
    typedef typename gsv_t<params>::instance_t instance_t;

    instance_t *d_instances;
    int32_t *d_results;
    cgbn_error_report_t *report;

    CUDA_CHECK(hipMalloc((void **)&d_instances, sizeof(instance_t) * MAX_INS));
    CUDA_CHECK(hipMalloc((void **)&d_results, sizeof(int32_t) * MAX_INS));
    CUDA_CHECK(cgbn_error_report_alloc(&report));

    test_sig_verify<params>(256, d_instances, d_results, report);

    // test_sig_verify<params>(32768, d_instances, d_results, report);

    for (int ins = 256; ins <= MAX_INS; ins *= 2) {
        printf("#instances: %d\n", ins);
        test_sig_verify<params>(ins, d_instances, d_results, report);
    }

    CUDA_CHECK(hipFree(d_instances));
    CUDA_CHECK(hipFree(d_results));
    CUDA_CHECK(cgbn_error_report_free(report));

    return 0;
}
